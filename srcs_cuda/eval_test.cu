#include "hip/hip_runtime.h"
#include "head.h"

int	count_moves(char *board, char player)
{
	int	i;
	int	out;

	i = 0;
	out = 0;
	while (i < 9)
	{
		out = out + (board[i] == player);
		i++;
	}
	return (out);
}

int	is_possible(char *board)
{
	return (count_moves(board, 'X') - count_moves(board, 'O') == 1 || count_moves(board, 'X') == count_moves(board, 'O'));
}

int	next_state(char *board, int pos)
{
//	printf("pos: %d\n", pos);
	if (pos == -1)
		return (0);
	if (board[pos] == '.')
		board[pos] = 'X';
	else if (board[pos] == 'X')
		board[pos] = 'O';
	else if (board[pos] == 'O')
	{
		board[pos] = '.';
		return (next_state(board, pos - 1));
	}
	return (1);

}

int		possible_states(void)
{
	char	*board;
	int		state;
	int		count;

	count = 0;
	board = init_board();
    db(smallest_value(board))[1] += 1;
	while (next_state(board, 8))
	{
		if (is_possible(board))
		{
			state = smallest_value(board);
   			db(state)[1] += 1;
			count++;
		}
	}
	free(board);
	return (count);
}

int		cardinal_state_space(void)
{
	possible_states();
	return (count_non_zeroes_db());
}

//does not assume X starts
int	is_possible_2(char *board)
{
	return (abs(count_moves(board, 'X') - count_moves(board, 'O')) == 1 || count_moves(board, 'X') == count_moves(board, 'O'));
}

int		possible_states_2(void)
{
	char	*board;
	int		count;

	count = 0;
	board = init_board();
	while (next_state(board, 8))
	{
		if (is_possible_2(board))
			count++;
	}
	free(board);
	return (count);
}

void print_state_space_info(void)
{
	possible_states();
	printf("total states: %d\nassuming X starts: %d \nstates after reduction %d\n", possible_states_2(), possible_states(), cardinal_state_space());
}


