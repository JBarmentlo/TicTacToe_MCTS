#include "hip/hip_runtime.h"
#include "head.h"
#include "maths.h"

int     base_conv(char c)
{
	if (c == PLAY_NONE)
		return (0);
	if (c == PLAY_ONE)
		return (1);
	if (c == PLAY_TWO)
		return (2);
	return (-1);
}

// assigns a number to a board considering it a ternary base number (cf base conv)
// the tranfo is a rotation or symmetry, the board is evaluated after the tranformation
// basically a base 3 atoi if that makes sense to you
int		trinary_board(char *board, char *tranfo)
{
	int		result = 0;
	int		i = -1;

	while (++i < 9)
	{
		result *= 3;
		result += base_conv(board[tranfo[i] - '0']);
	}
	return (result);
}

__device__ int	base_conv_cuda(char c) //ALWAYS INLINE
{
	if (c == PLAY_NONE)
		return (0);
	if (c == PLAY_ONE)
		return (1);
	if (c == PLAY_TWO)
		return (2);
	return (-1);
}

__global__ void	trinary_board_cuda(char *board, char *transfo, int *out)
{
	int		result = 0;
	int		i = 0;

	while (i < 9)
	{
		result *= 3;
		result += base_conv_cuda(board[transfo[(threadIdx.x * 9) + i] - '0']);
		i++;
	}
	out[threadIdx.x] = result;
}

int     smallest_value(char *board)
{
	static int mallocced = 1;
	int i;
	char *transformations[] = transformationes;
	static char *transformations_d;
	static char *board_d;
	static int	*val_d;
	static int	*val;
	int out;
//	printf("1\n");

	if (mallocced)
	{
		printf("MALLOCCING\n");
		val = (int*)malloc(8 * sizeof(int));
		hipMalloc(&val_d, 8 * sizeof(int));
		hipMalloc(&transformations_d, 8 * 9);
		hipMalloc(&board_d, 9);

		i = 0;
		while (i < 8)
		{
			hipMemcpy(&transformations_d[9 * i], transformations[i], 9, hipMemcpyHostToDevice);	
			i++;
		}
		mallocced = 0;
	}
	hipMemcpy(board_d, board, 9, hipMemcpyHostToDevice);

	trinary_board_cuda<<<1,8>>>(board_d, transformations_d, val_d);
	hipMemcpy(val, val_d, 8 * sizeof(int), hipMemcpyDeviceToHost);

	out = val[0];
	i = 1;
	while (i < 8)
	{
		if (val[i] < out)
		{
			out = val[i];
		}
		i++;
	}
    return (out);
}

// looks for the transformation resulting in the smallest output for trinary_board and returns that value;
int     smallest_value_tmp(char *board)
{
    int result;
    int tampoun;
    int i;
    char *transformations[] = transformationes;

    result = trinary_board(board, identity);
    i = 1;
    while (i < 8)
    {
        tampoun = trinary_board(board, transformations[i]);
        if (tampoun < result)
            result = tampoun;
        i++;
    }
    return (result);
}


//will print the combination of two transformations on terminal.
void    print_rots(char *first, char *second)
{
    int i;

    i = 0;
    while (i < 9)
    {
        printf("%c",first[second[i] - 48]);
        i++;
    }
}